#include "utils.cuh"

void printCudaDeviceProperties(hipDeviceProp_t& deviceProp) {
	cout << "===========================================================" << endl;
	cout << "Device " << deviceProp.name << " Properties:" << endl;
	cout << "Clock Rate (KHz) " << deviceProp.clockRate << endl;
	cout << "Memory Clock Rate (KHz) " << deviceProp.memoryClockRate << endl;
	cout << "Total Global Memory (MB):\t" << (deviceProp.totalGlobalMem / MB) << endl;
	cout << "Shared Memory / Block (KB):\t" << (deviceProp.sharedMemPerBlock / KB) << endl;
	// Warp: groups of threads that executes the same instruction
	cout << "Warp Size:\t" << deviceProp.warpSize << endl;
	// Pitch: Padded size of each row in an array?
	cout << "Pitch (MB):\t" << (deviceProp.memPitch / MB) << endl;
	cout << "Max Threads / Block:\t" << deviceProp.maxThreadsPerBlock << endl;
	cout << "Max Dimension Size of Block :\t" << deviceProp.maxThreadsDim[0] << ", " << deviceProp.maxThreadsDim[1] << ", " << deviceProp.maxThreadsDim[2] << endl;
	cout << "Max Dimension Size of Grid :\t" << deviceProp.maxGridSize[0] << ", " << deviceProp.maxGridSize[1] << ", " << deviceProp.maxGridSize[2] << endl;
	cout << "Multiprocess Count:\t" << deviceProp.multiProcessorCount << endl;
	cout << "Max Blocks / Multiprocessor:\t" << deviceProp.maxBlocksPerMultiProcessor << endl;
	cout << "Concurrent Kernels:\t" << deviceProp.concurrentKernels << endl;
	cout << "Max Threads / Multiprocessor:\t" << deviceProp.maxThreadsPerMultiProcessor << endl;
	cout << "Shared Memory (KB) / Multiprocessor:\t" << (deviceProp.sharedMemPerMultiprocessor / KB) << endl;
	cout << "===========================================================" << endl;
}
