#include <iostream>
#include <filesystem>
#include <chrono> // For timing
#include <vector> // For storing execution times
#include <numeric> // For calculating averages
#include "utils/mnist_dataloader.h"
#include "utils/runner.cuh"
#include "utils/utils.cuh"
#include "kernels/kernels.cuh"
#include "hip/hip_runtime.h"

const uint8_t DEFAULT_IMAGE_WIDTH = 28; // MNIST images are 28x28
const uint8_t DEFAULT_IMAGE_HEIGTH = 28;


// [Note from g.agluba to all],
// Please check Runner / Kernel Naming convention
// Reference first:
//      Optimizations:
//          class 0: shared memory and tiling ?
//          class 1: warping
//          class 3. loop unrolling
//          class 4. parallel scan (according to some research, this is feasible  
//      ....  
//  THINK OF ANYTHING
// 
// Assuming we have 6 optimization (we can always change this),
//      K000000 means unoptimized code running based kernel kmeans_000000
//      K100000 means optimized kernel with shared memory kmeans_100000, kmeans_X00000 X>1 for any other variants
//      K101000 mean optimized kernel with shared memory and loop unrolling
//      ...


class K000000Runner : public BaseRunner {
public:
    void runKernel(
        dim3 dimGrid,
        dim3 dimBlock,
        // [g.agluba note] 
        // typically use float, but since most data are positive int, will make unsigned int to reduce memory
        uint8_t* images_d, //flatten images of size N X IMAGE_HEIGHT X IMAGE_WIDTH
        size_t N, // number of images = 6000?
        uint8_t IMAGE_HEIGHT, // image height = 28
        uint8_t IMAGE_WIDTH, // image width = 28
        uint8_t* K_cluster_d, // array to hold K cluster label
        uint8_t K, // k-means parameter
        float* centroids_d, // flatten centroids of size K X IMAGE_HEIGHT X IMAGE_WIDTH
        int max_iter
    ) {
        kmeans_000000 << <dimGrid, dimBlock >> > (
            images_d,
            N,
            IMAGE_HEIGHT,
            IMAGE_WIDTH,
            K_cluster_d,
            K,
            centroids_d,
            max_iter
        );
    }
};

// ADD Extended class here for other kernels
class K100000Runner : public BaseRunner {
public:
    void runKernel(
        dim3 dimGrid,
        dim3 dimBlock,
        uint8_t* images_d,
        size_t N,
        uint8_t IMAGE_HEIGHT,
        uint8_t IMAGE_WIDTH,
        uint8_t* K_cluster_d,
        uint8_t K,
        float* centroids_d,
        int max_iter
    ) {
        // Calculate shared memory size for centroids
        size_t sharedMemorySize = K * IMAGE_HEIGHT * IMAGE_WIDTH * sizeof(float);

        // Launch the kmeans_100000 kernel
        kmeans_100000<<<dimGrid, dimBlock, sharedMemorySize>>>(
            images_d,
            N,
            IMAGE_HEIGHT,
            IMAGE_WIDTH,
            K_cluster_d,
            K,
            centroids_d,
            max_iter
        );
    }
};


int main() {
    try {
        // Device Properties
        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        std::cout << "Device Count\t" << deviceCount << std::endl;

        for (int device = 0; device < deviceCount; device++) {
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, device);
            printCudaDeviceProperties(deviceProp);
        }

        // Load Data
        MNISTDataLoader loader("../data/MNIST/raw/train-images-idx3-ubyte", "../data/MNIST/raw/train-labels-idx1-ubyte");
        loader.load();
        const auto& images = loader.getImages();
        const auto& labels = loader.getLabels();
        std::cout << "Loaded " << images.size() << " images and " << labels.size() << " labels." << std::endl;
        loader.visImg(3456); // visualize one image given data index, comment if you want

        // [todo g.agluba]
        // get command-line arguments for easier testing ... 
        // for now, edit this when testing

        // // Run kmeans_000000
        // K000000Runner runner000000 = K000000Runner();
        // runner000000.run(images, images.size(), DEFAULT_IMAGE_HEIGTH, DEFAULT_IMAGE_WIDTH, labels);

        // // Run kmeans_100000
        // K100000Runner runner100000 = K100000Runner();
        // runner100000.run(images, images.size(), DEFAULT_IMAGE_HEIGTH, DEFAULT_IMAGE_WIDTH, labels);

        // Warm-Up Step
        std::cout << "Running warm-up..." << std::endl;
        K100000Runner warmupRunner = K100000Runner();
        warmupRunner.run(images, images.size(), DEFAULT_IMAGE_HEIGTH, DEFAULT_IMAGE_WIDTH, labels);
        hipDeviceSynchronize(); // Ensure kernel execution is complete
        std::cout << "Warm-up completed." << std::endl;

        // Timing variables
        std::vector<double> execution_times;

        // Run K100000Runner 5 times
        for (int i = 0; i < 5; ++i) {
            K100000Runner runner = K100000Runner();
            auto start = std::chrono::high_resolution_clock::now();
            runner.run(images, images.size(), DEFAULT_IMAGE_HEIGTH, DEFAULT_IMAGE_WIDTH, labels);
            hipDeviceSynchronize(); // Ensure kernel execution is complete
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double> elapsed = end - start;
            execution_times.push_back(elapsed.count());
            std::cout << "K100000Runner Run " << i + 1 << ": " << elapsed.count() << " seconds" << std::endl;
        }

        // Calculate and display average execution time
        double average_time = std::accumulate(execution_times.begin(), execution_times.end(), 0.0) / execution_times.size();
        std::cout << "\nAverage Execution Time for K100000Runner: " << average_time << " seconds" << std::endl;

    } catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
        return 1;
    }

    return 0;
}
